#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include "utils.h"

__global__ void print_idx_kernel() {
    printf("block idx: (%3d, %3d, %3d), thread idx: (%3d, %3d, %3d)\n",
        blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ void print_dim_kernel() {
    printf("grid dimension: (%3d, %3d, %3d), thread dimension: (%3d, %3d, %3d)\n",
        gridDim.z, gridDim.y, gridDim.x, blockDim.z, blockDim.y, blockDim.x);
}

__global__ void print_thread_idx_per_block_kernel() { 
    int index = threadIdx.z * blockDim.x * blockDim.y + \
                threadIdx.y * blockDim.x + \
                threadIdx.x;
    printf("block idx: (%3d, %3d, %3d), thread idx: %3d\n",
        blockIdx.z, blockIdx.y, blockIdx.x, index);
}

__global__ void print_thread_idx_per_grid_kernel() {
    int bSize = blockDim.z * blockDim.y * blockDim.z;
    
    int bIndex = blockIdx.z * gridDim.x * gridDim.y + \
                 blockIdx.y * gridDim.x + \
                 blockIdx.x;

    int tIndex = threadIdx.z * blockDim.x * blockDim.y + \
                 threadIdx.y * blockDim.x + \
                 threadIdx.x;
    
    int index = bIndex * bSize + tIndex;
    
    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n", bIndex, tIndex, index);
}

__global__ void print_cord_kernel() {
    int index = threadIdx.z * blockDim.x * blockDim.y + \
                threadIdx.y * blockDim.x + \
                threadIdx.x;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord:(%3d, %3d)\n", blockIdx.z, blockIdx.y, blockIdx.x, index, x, y);
}

void print_one_dim() {
    int inputSize = 8;
    int blockDim = 4;
    int gridDim = inputSize / blockDim;
    
    dim3 block(blockDim);
    dim3 grid(gridDim);

    print_idx_kernel<<<grid, block>>>();
    print_dim_kernel<<<grid, block>>>();
    print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    CUDA_CHECK(hipDeviceSynchronize());
}

void print_two_dim() {
    int inputSize = 8;
    int blockDim = 2;
    int gridDim = inputSize / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_idx_kernel<<<grid, block>>>();
    print_dim_kernel<<<grid, block>>>();
    print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    CUDA_CHECK(hipDeviceSynchronize());
}

void print_cord() {
    int inputWidth = 4;
    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_cord_kernel<<<grid, block>>>();

    CUDA_CHECK(hipDeviceSynchronize());
}


int main(void) {  
	printf("Hello CUDA from host!\n"); 
  
	print_one_dim();
    // print_two_dim();
    // print_cord();
    return 0; 
}

